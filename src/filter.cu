#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define BLOCK_SIZE 128

__global__ void av3(int n, float *in1, float *in2, float *in3, float *out)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index<n)
	{
		out[index] = (in1[index] + in2[index] + in3[index])/3;
	}
}

float *average3(int num, float *in1, float *in2, float *in3, float *out) 
{
	// Initialization
	float *d_in1, *d_in2, *d_in3, *d_out; 
	int size = num * sizeof(float);

	// Allocate memory for device
	hipMalloc((void **)&d_in1, size);
	hipMalloc((void **)&d_in2, size);
	hipMalloc((void **)&d_in3, size);
	hipMalloc((void **)&d_out, size);

	// Copy inputs to device
	hipMemcpy(d_in1, in1, size, hipMemcpyHostToDevice);
	hipMemcpy(d_in2, in2, size, hipMemcpyHostToDevice);
	hipMemcpy(d_in3, in3, size, hipMemcpyHostToDevice);

	// Launch Kernel
	dim3 DimGrid((num-1)/BLOCK_SIZE+1, 1, 1);
	dim3 DimBlock(BLOCK_SIZE, 1, 1);
	av3<<<DimGrid,DimBlock>>>(num, d_in1, d_in2, d_in3, d_out);

	// Wait for the GPU to finish
	hipDeviceSynchronize();

	// Copy result back to host and cleanup
	hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
	hipFree(d_in1); hipFree(d_in2); hipFree(d_in3); hipFree(d_out);
	return out;
}
